/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* matrix project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

#include <matrix.h>
#include <cudaCom.h>

// includes, kernels
#include <matrix_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold(float* matrixA, 
				 float* matrixB, 
				 float* matrixC, 
				 int ah,
				 int aw,
				 int bh,
				 int bw);


void printMatrix(float* matrix, const unsigned int width, const unsigned int  matrixsize) 
{
//	if (1>0) return;
	for (unsigned int i=0; i<matrixsize; i++)
	{
		printf(" %f\t ", matrix[i]);
		if ((i+1)%width==0) 
		{
			printf("\n");
		}
	}
	printf("\n");
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f \n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{

    CUT_DEVICE_INIT(argc, argv);


	unsigned int matrixASize = MATRIX_A_WIDTH*MATRIX_A_HEIGHT;
    unsigned int matrixBSize = MATRIX_B_WIDTH*MATRIX_B_HEIGHT;
    unsigned int matrixCSize = matrixASize;

    unsigned int matrixMemASize = sizeof( float) * MATRIX_A_WIDTH*MATRIX_A_HEIGHT;
    unsigned int matrixMemBSize = sizeof( float) * MATRIX_B_WIDTH*MATRIX_B_HEIGHT;
    unsigned int matrixMemCSize = matrixMemASize;

    // allocate host memory
    float* matrixA = (float*) malloc( matrixMemASize);
    float* matrixB = (float*) malloc( matrixMemBSize);
    float* matrixC = (float*) malloc( matrixMemCSize);
//    unsigned int num_threads = matrixCSize;

	printf( "initalize A\n");
    // initalize the memory
#ifndef USE_TEST_INPUTS
// Original inputs given
    for( unsigned int i = 0; i < matrixASize; ++i) matrixA[i] = (float) i;
    for( unsigned int i = 0; i < matrixBSize; ++i) matrixB[i] = (float) (i+1);
#else
//Test inputs given
    for( unsigned int i = 0; i < matrixASize; ++i) matrixA[i] = (float) 1;
    for( unsigned int i = 0; i < matrixBSize; ++i) matrixB[i] = (float) 1;
#endif
	// print matrix 
	printf( "matrix A\n");
//	printMatrix(matrixA, MATRIX_A_WIDTH, matrixASize);

	printf( "matrix B\n");
//	printMatrix(matrixB, MATRIX_B_WIDTH, matrixBSize);

	// allocate device memory
    float* d_matrixA;
    float* d_matrixB;
    float* d_matrixC;

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_matrixA, matrixMemASize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_matrixB, matrixMemBSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_matrixC, matrixMemCSize));


    timestamp("start", 0);
	
	// copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_matrixA, matrixA, matrixMemASize,
                                hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( d_matrixB, matrixB, matrixMemBSize,
                                hipMemcpyHostToDevice) );
//    CUDA_SAFE_CALL( hipMemcpy( d_matrixC, matrixC, matrixCSize,
//                                hipMemcpyHostToDevice) );

    // allocate device memory for result
#if 0
// No Optimization
    // setup execution parameters
    dim3  threads( BLOCK_SIZE, BLOCK_SIZE);
    dim3  grid( MATRIX_A_WIDTH/BLOCK_SIZE, MATRIX_A_HEIGHT/BLOCK_SIZE);
#else
    // setup execution parameters
    dim3  threads( BLOCK_SIZE_WIDTH, BLOCK_SIZE_HEIGHT);
    dim3  grid( (MATRIX_A_WIDTH)/BLOCK_SIZE_WIDTH, MATRIX_A_HEIGHT/2);
#endif // No Optimization

    // execute the kernel
    testKernel<<< grid, threads >>>( d_matrixA, d_matrixB, d_matrixC, MATRIX_A_HEIGHT, MATRIX_A_WIDTH, MATRIX_B_HEIGHT, MATRIX_B_WIDTH);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( matrixC, d_matrixC, matrixMemCSize,
                                hipMemcpyDeviceToHost) );

    timestamp("done kernel", 0);
//	printf( "matrixC\n");
//	printMatrix(matrixC, MATRIX_A_WIDTH, matrixCSize);

    // compute reference solution
    float* reference = (float*) malloc( matrixMemCSize);
	computeGold(matrixA, matrixB, reference, MATRIX_A_HEIGHT, MATRIX_A_WIDTH, MATRIX_B_HEIGHT, MATRIX_B_WIDTH);
//	printf( "reference\n");
//	printMatrix(reference, MATRIX_A_WIDTH, matrixCSize);

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
//        CUT_SAFE_CALL( cutWriteFilef( "./data/regression.dat",
//                                      h_odata, num_threads, 0.0));
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        int res = compMatrix( reference, matrixC, matrixCSize);
        printf( "Test %s\n", (1 == res) ? "PASSED" : "FAILED");
        if (res!=1) printDiff(reference, matrixC, MATRIX_A_WIDTH, MATRIX_A_HEIGHT);
    }

    // cleanup memory
    free( matrixA);
    free( matrixB);
    free( matrixC);
    free( reference);
    CUDA_SAFE_CALL(hipFree(d_matrixA));
    CUDA_SAFE_CALL(hipFree(d_matrixB));
    CUDA_SAFE_CALL(hipFree(d_matrixC));
}
