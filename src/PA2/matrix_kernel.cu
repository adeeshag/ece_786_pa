#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* matrix project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code.
 */

#ifndef _matrix_KERNEL_H_
#define _matrix_KERNEL_H_

#define KERNEL_SIZE 64
#define KERNEL_LENGTH 8
#define WARP_SIZE 32
#define FACTOR	1 
#define DATA_TO_PULL_SIZE (FACTOR * WARP_SIZE)

//#define CHANGE1 1
#define CHANGE2 1

#include <stdio.h>

#include <matrix.h>

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(	float* d_matrixA, 
			float* d_matrixB, 
			float* d_matrixC,
	   		const unsigned int ah,
			const unsigned int aw,
			const unsigned int bh,
			const unsigned int bw) {
  // shared memory
#ifdef CHANGE1

    unsigned int shm_matrixBSize = sizeof(float) * bh * bw ; //Matrix B size
    __shared__ float shm_matrixB[KERNEL_SIZE];
#endif
#ifdef CHANGE2
    __shared__ float shm_subMatrixA[BLOCK_SIZE_WIDTH];

#endif
  // the size is determined by the host application
// extern  __shared__  float sdata[];
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;


	// access thread id
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

#if 1
	int xstep = bx;
	int ystep = BLOCK_SIZE_HEIGHT*by;

#endif

	float sum = 0;
#if 1
	int y = ystep+ty;
	int x = xstep+ tx;
#endif


#ifdef CHANGE1

	if((tx<8)&&(ty<8))
	    shm_matrixB[KERNEL_LENGTH * ty + tx] = d_matrixB[ ty * KERNEL_LENGTH + tx];

	__syncthreads();
#endif


#ifdef CHANGE2	
//modified code
	for (int j=0; j<bh; j++) {

#if 0
			if(tx<WARP_SIZE)
		           if (((y-j)>-1) &&((y-j)<ah)&&((x-DATA_TO_PULL_SIZE)>-1)&&((x - DATA_TO_PULL_SIZE)<aw)) 
		   	      shm_subMatrixA[tx] = d_matrixA[(y-j)*aw+(x-DATA_TO_PULL_SIZE)];
#endif
#if 1	

		        if ((((y-j)>-1) &&(y-j)<ah))
		   	   shm_subMatrixA[tx] = d_matrixA[(y-j)*aw+(x)];
#endif
		   
		__syncthreads();

		for(int k = 0; k < bw; ++k) {
			float b = d_matrixB[j*bw+k];
			float a = 0;
			// check the out-of-bound
			if ((y-j)>-1 &&(y-j)<ah&&((x)-k)>-1&&((x)-k)<aw) {
				    a = shm_subMatrixA[tx-k];

				sum += a*b;
			}
		}//k loop
		__syncthreads();
	}//j loop
#elif defined(CHANGE1)
//modified code
	for (int j=0; j<bh; j++) {
		for(int k = 0; k < bw; ++k) {
			float b = shm_matrixB[j*bw+k];
			float a = 0;
			// check the out-of-bound
			if ((y-j)>-1&&(y-j)<ah&&(x-k)>-1&&(x-k)<aw) {
				a = d_matrixA[(y-j)*aw+(x-k)];
				sum += a*b;
			}
		}
	} //j loop
	__syncthreads();
#else
//Original Code
	for (int j=0; j<bh; j++) {
		for(int k = 0; k < bw; ++k) {
			float b = d_matrixB[j*bw+k];
			float a = 0;
			// check the out-of-bound
			if ((y-j)>-1&&(y-j)<ah&&(x-k)>-1&&(x-k)<aw) {
				a = d_matrixA[(y-j)*aw+(x-k)];
				sum += a*b;
			}
		}
	}//j loop
#endif
	// write data to global memory
	d_matrixC[y*aw+x] = sum;
}// end of func

#endif // #ifndef _matrix_KERNEL_H_
