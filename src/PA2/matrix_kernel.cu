#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* matrix project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code.
 */

#ifndef _matrix_KERNEL_H_
#define _matrix_KERNEL_H_

#include <stdio.h>

#include <matrix.h>

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(	float* d_matrixA, 
			float* d_matrixB, 
			float* d_matrixC,
	   		const unsigned int ah,
			const unsigned int aw,
			const unsigned int bh,
			const unsigned int bw) {
  // shared memory
    unsigned int matrixBSize = bh*bw; //Matrix B size
	__shared__ float shm_matrixB[];
  // the size is determined by the host application
// extern  __shared__  float sdata[];
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;


	// access thread id
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	int xstep = BLOCK_SIZE*bx;
	int ystep = BLOCK_SIZE*by;


	float sum = 0;
	int y = ystep+ty;
	int x = xstep+tx;
	
	for (int j=0; j<bh; j++) {
		for(int k = 0; k < bw; ++k) {
			float b = d_matrixB[j*bw+k];
			float a = 0;
			// check the out-of-bound
			if ((y-j)>-1&&(y-j)<ah&&(x-k)>-1&&(x-k)<aw) {
				a = d_matrixA[(y-j)*aw+(x-k)];
				sum += a*b;
			}
		}
	}

	// write data to global memory
	d_matrixC[y*aw+x] = sum;
}

#endif // #ifndef _matrix_KERNEL_H_
